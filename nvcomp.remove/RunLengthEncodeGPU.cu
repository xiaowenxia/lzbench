#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "RunLengthEncodeGPU.h"
#include "TempSpaceBroker.h"
#include "common.h"
#include "CascadedCommon.h"
#include "nvcomp.hpp"
#include "type_macros.h"

#ifdef __GNUC__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Weffc++"
#pragma GCC diagnostic ignored "-Wunused-parameter"
#endif
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>
#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif

#include <cassert>
#include <stdexcept>
#include <string>

namespace nvcomp
{

/******************************************************************************
 * CONSTANTS ******************************************************************
 *****************************************************************************/

namespace
{
constexpr const size_t ALIGN_OFFSET = 256;
constexpr const int WARP_SIZE = 32;
constexpr const int GLOBAL_TILE_SIZE = 1024;
} // namespace

/******************************************************************************
 * KENRELS ********************************************************************
 *****************************************************************************/

namespace
{

template <typename T, int NUM_THREADS>
__device__ T warpSum(T const initVal)
{
  constexpr const uint32_t mask
      = NUM_THREADS < WARP_SIZE ? (1u << NUM_THREADS) - 1 : 0xffffffff;
  T val = initVal;
  for (int d = NUM_THREADS / 2; d > 0; d /= 2) {
    val += __shfl_down_sync(mask, val, d, NUM_THREADS);
  }

  return val;
}

template <typename T, int BLOCK_SIZE>
__device__ T cooperativeSum(T const initVal, T* const buffer)
{
  // first all warps reduce to single value
  assert(BLOCK_SIZE % WARP_SIZE == 0);
  assert(BLOCK_SIZE <= WARP_SIZE * WARP_SIZE);

  T val = warpSum<T, WARP_SIZE>(initVal);
  if (threadIdx.x % WARP_SIZE == 0) {
    buffer[threadIdx.x / WARP_SIZE] = val;
  }
  __syncthreads();

  if (threadIdx.x < (BLOCK_SIZE / WARP_SIZE)) {
    val = warpSum<T, BLOCK_SIZE / WARP_SIZE>(buffer[threadIdx.x]);
  }

  return val;
}

/**
 * @brief This kernel produces the block sizes for a prefixsum in a subsequent
 * kernel.
 *
 * @tparam VALUE The value type.
 * @tparam RUN The run count type.
 * @param in The input data.
 * @param num The size of the input data.
 * @param blockSize The location to write the block sizes (output).
 */
template <typename VALUE, typename RUN, int BLOCK_SIZE, int TILE_SIZE>
__global__ void rleInitKernel(
    const VALUE* const in,
    const size_t* const numInDevice,
    RUN* const blockSize)
{
  constexpr const int ITEMS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;
  // the algorithm here is to keep reducing "chunks" to a start and end marker

  const int num = static_cast<int>(*numInDevice);

  if (blockIdx.x * TILE_SIZE < num) {
    // we load the preceding value in the first spot
    __shared__ VALUE valBuffer[TILE_SIZE + 1];
    __shared__ RUN buffer[BLOCK_SIZE / WARP_SIZE];

    if (threadIdx.x == 0) {
      valBuffer[0]
          = blockIdx.x > 0 ? in[blockIdx.x * TILE_SIZE - 1] : (in[0] + 1);
    }
    for (int tid = threadIdx.x; tid < TILE_SIZE; tid += BLOCK_SIZE) {
      const int gTid = tid + blockIdx.x * TILE_SIZE;
      // cooperatively populate valBuffer and runBuffer
      if (gTid < num) {
        valBuffer[tid + 1] = in[gTid];
      } else {
        valBuffer[tid + 1] = in[num - 1];
      }
    }

    __syncthreads();

    // build bit mask
    VALUE val = valBuffer[threadIdx.x * ITEMS_PER_THREAD];
    RUN sum = 0;
    for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
      const int tid = threadIdx.x * ITEMS_PER_THREAD + i;
      const VALUE nextVal = valBuffer[tid + 1];
      sum += nextVal != val;
      val = nextVal;
    }

    sum = cooperativeSum<RUN, BLOCK_SIZE>(sum, buffer);
    if (threadIdx.x == 0) {
      blockSize[blockIdx.x] = sum;
    }
  } else if (threadIdx.x == 0) {
    blockSize[blockIdx.x] = 0;
  }

  if (blockIdx.x == gridDim.x - 1 && threadIdx.x == 0) {
    blockSize[gridDim.x] = 0;
  }
}

template <typename VALUE, typename RUN, int BLOCK_SIZE, int TILE_SIZE>
__global__ void rleReduceKernel(
    const VALUE* const in,
    const size_t* const numInDevice,
    const RUN* const blockPrefix,
    RUN* const blockStart,
    VALUE** const valsPtr,
    RUN** const runsPtr,
    size_t* const numOutDevice)
{
  constexpr const int ITEMS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;
  // the algorithm here is to keep reducing "chunks" to a start and end marker
  const int num = static_cast<int>(*numInDevice);

  if (blockIdx.x * TILE_SIZE < num) {
    VALUE* const vals = *valsPtr;
    RUN* const runs = *runsPtr;

    // we load the preceding value in the first spot
    __shared__ VALUE valBuffer[TILE_SIZE + 1];

    // we store the sum in the last spot
    __shared__ RUN prefix[BLOCK_SIZE + 1];

    if (threadIdx.x == 0) {
      valBuffer[0]
          = blockIdx.x > 0 ? in[blockIdx.x * TILE_SIZE - 1] : (in[0] + 1);
    }
    for (int tid = threadIdx.x; tid < TILE_SIZE; tid += BLOCK_SIZE) {
      const int gTid = tid + blockIdx.x * TILE_SIZE;
      // cooperatively populate valBuffer and runBuffer
      if (gTid < num) {
        valBuffer[tid + 1] = in[gTid];
      } else {
        valBuffer[tid + 1] = in[num - 1];
      }
    }

    __syncthreads();

    // build bit mask
    RUN sum = 0;
    {
      VALUE val = valBuffer[threadIdx.x * ITEMS_PER_THREAD];
      for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
        const int tid = threadIdx.x * ITEMS_PER_THREAD + i;
        const VALUE nextVal = valBuffer[tid + 1];
        sum += nextVal != val;
        val = nextVal;
      }
    }

    __syncthreads();

    // prefixsum bit mask
    {
      typedef hipcub::BlockScan<RUN, BLOCK_SIZE> BlockScan;
      __shared__ typename BlockScan::TempStorage temp_storage;

      BlockScan(temp_storage).ExclusiveSum(sum, sum);

      prefix[threadIdx.x] = sum;
      if (threadIdx.x == 0) {
        prefix[BLOCK_SIZE]
            = blockPrefix[blockIdx.x + 1] - blockPrefix[blockIdx.x];
      }
    }

    __syncthreads();

    __shared__ RUN runBuffer[TILE_SIZE + 1];

    // do local run length encoding with an undifferentiated run count
    {
      int outIdx = prefix[threadIdx.x];
      VALUE val = valBuffer[threadIdx.x * ITEMS_PER_THREAD];
      for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
        const int tid = threadIdx.x * ITEMS_PER_THREAD + i;
        const VALUE nextVal = valBuffer[tid + 1];
        if (nextVal != val) {
          runBuffer[outIdx] = tid;

          val = nextVal;
          ++outIdx;
        }
      }
    }

    const RUN numCompacted = prefix[BLOCK_SIZE];
    if (threadIdx.x == 0) {
      runBuffer[numCompacted] = ((blockIdx.x + 1) * TILE_SIZE >= num)
                                    ? ((num - 1) % TILE_SIZE) + 1
                                    : TILE_SIZE;
    }

    __syncthreads();

    // write back to global memory
    const RUN offset = blockPrefix[blockIdx.x];
    for (int tid = threadIdx.x; tid < numCompacted; tid += BLOCK_SIZE) {
      // runs still need to be differentiated -- the last one will need to the
      // number of values
      vals[offset + tid] = valBuffer[runBuffer[tid] + 1];
      assert(runBuffer[tid + 1] >= runBuffer[tid]);
      runs[offset + tid] = runBuffer[tid + 1] - runBuffer[tid];
    }

    if (threadIdx.x == 0) {
      blockStart[blockIdx.x] = runBuffer[0] + blockIdx.x * TILE_SIZE;
    }
  }
  if (blockIdx.x == gridDim.x - 1 && threadIdx.x == BLOCK_SIZE - 1) {
    *numOutDevice = blockPrefix[gridDim.x];
  }
}

/**
 * @brief Fix block join gaps, that is where the run count for a given number
 * fails to account for duplicates in the following block(s). This requires
 * that the first run in each block's output, not be differentiated.
 *
 * @param runs The almost finished runs.
 * @param blockPrefix The previously calculated block prefix.
 * @param num The number of entries.
 */
template <typename RUN, int BLOCK_SIZE, int TILE_SIZE>
__global__ void rleFinalizeKernel(
    RUN** const runsPtr,
    const RUN* const blockStart,
    const RUN* const blockPrefix,
    const size_t* const numInDevice)
{
  const int num = roundUpDiv(static_cast<int>(*numInDevice), TILE_SIZE);

  if (blockIdx.x * BLOCK_SIZE < num) {
    RUN* const runs = *runsPtr;

    // we load the blocks runs plus 1 extra
    __shared__ RUN prefixBuffer[BLOCK_SIZE + 1];

    int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;

    const RUN bp = tid < num ? blockPrefix[tid] : blockPrefix[num];
    prefixBuffer[threadIdx.x] = bp;

    if (threadIdx.x == 0) {
      prefixBuffer[BLOCK_SIZE] = blockPrefix[(blockIdx.x + 1) * BLOCK_SIZE];
    }

    __syncthreads();

    if (tid < num) {
      if (bp > 0 && (tid + 1 == num || bp < prefixBuffer[threadIdx.x + 1])) {
        // TODO: make this a binary search

        int low = 0;
        int high = tid;
        while (high > low) {
          const int mid = (low + high) / 2;
          if (blockPrefix[mid] == bp) {
            // keep searching down
            high = mid;
          } else {
            // start searching up
            low = mid + 1;
          }
        }

        // we need to fix the count for this block
        runs[bp - 1] += blockStart[tid] - low * TILE_SIZE;
      }
    }
  }
}
} // namespace

/******************************************************************************
 * HELPER FUNCTIONS ***********************************************************
 *****************************************************************************/

namespace
{

template <typename T>
size_t downstreamWorkspaceSize(const size_t num)
{
  return sizeof(T) * std::max(1024ULL, 3ULL * roundUpDiv(num, GLOBAL_TILE_SIZE))
         + sizeof(int);
}

template <typename T, typename U>
size_t requiredWorkspaceSizeTyped(const size_t num)
{
  // TODO: this assume large datatype
  T* inPtr = nullptr;
  T* valsPtr = nullptr;
  U* runsPtr = nullptr;
  size_t* numPtr = nullptr;

  size_t workspaceSize = 0;
  hipError_t err = hipcub::DeviceRunLengthEncode::Encode(
      nullptr,
      workspaceSize,
      inPtr,
      valsPtr,
      runsPtr,
      numPtr,
      static_cast<int>(num),
      0);
  if (err != hipSuccess) {
    throw std::runtime_error(
        "Failed to get workspace size: " + std::to_string(err));
  }

  workspaceSize = std::max(workspaceSize, downstreamWorkspaceSize<U>(num));

  return ALIGN_OFFSET + workspaceSize;
}

template <typename VALUE, typename COUNT>
void compressInternal(
    void* const workspace,
    const size_t workspaceSize,
    void* const outValues,
    void* const outCounts,
    size_t* numOutDevice,
    void const* const in,
    size_t const num,
    hipStream_t stream)
{
  VALUE* const outValuesTyped = static_cast<VALUE*>(outValues);
  COUNT* const outCountsTyped = static_cast<COUNT*>(outCounts);
  const VALUE* const inTyped = static_cast<const VALUE*>(in);

  const size_t reqWorkspaceSize = RunLengthEncodeGPU::requiredWorkspaceSize(
      num, getnvcompType<VALUE>(), getnvcompType<COUNT>());
  if (workspaceSize < reqWorkspaceSize) {
    throw std::runtime_error(
        "Invalid workspace size: " + std::to_string(workspaceSize)
        + ", need at least " + std::to_string(reqWorkspaceSize));
  }

  void* const alignedWorkspace = align(workspace, ALIGN_OFFSET);
  size_t alignedWorkspaceSize
      = workspaceSize - relativeEndOffset(workspace, alignedWorkspace);

  hipError_t err = hipcub::DeviceRunLengthEncode::Encode(
      alignedWorkspace,
      alignedWorkspaceSize,
      inTyped,
      outValuesTyped,
      outCountsTyped,
      numOutDevice,
      static_cast<int>(num),
      stream);
  if (err != hipSuccess) {
    throw std::runtime_error(
        "Failed to schedule hipcub::DeviceRunLengthEncode::Encode() kernel: "
        + std::to_string(err));
  }
}

template <typename VALUE, typename COUNT>
void compressDownstreamInternal(
    void* const workspace,
    const size_t workspaceSize,
    void** const outValuesPtr,
    void** const outCountsPtr,
    size_t* numOutDevice,
    void const* const in,
    size_t const* numInDevice,
    const size_t maxNum,
    hipStream_t stream)
{
  VALUE** const outValuesTypedPtr = reinterpret_cast<VALUE**>(outValuesPtr);
  COUNT** const outCountsTypedPtr = reinterpret_cast<COUNT**>(outCountsPtr);
  const VALUE* const inTyped = static_cast<const VALUE*>(in);

  const size_t reqWorkspaceSize = downstreamWorkspaceSize<COUNT>(maxNum);
  if (workspaceSize < reqWorkspaceSize) {
    throw std::runtime_error(
        "Invalid workspace size: " + std::to_string(workspaceSize)
        + ", need at least " + std::to_string(reqWorkspaceSize));
  }

  constexpr const int BLOCK_SIZE = 128;

  const dim3 grid(roundUpDiv(maxNum, GLOBAL_TILE_SIZE));
  const dim3 block(BLOCK_SIZE);

  TempSpaceBroker tempSpace(workspace, workspaceSize);

  COUNT* blockSizes;
  COUNT* blockPrefix;
  COUNT* blockStart;
  tempSpace.reserve(&blockSizes, grid.x);
  tempSpace.reserve(&blockPrefix, grid.x + 1);
  tempSpace.reserve(&blockStart, grid.x);

  void* const scanWorkspace = tempSpace.next();

  // TODO: expand such that the mask calculation is done across the entire
  // array, and the the prefixsum, and then reduction

  // get blocks sizes
  rleInitKernel<VALUE, COUNT, BLOCK_SIZE, GLOBAL_TILE_SIZE>
      <<<grid, block, 0, stream>>>(inTyped, numInDevice, blockSizes);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(
        "Failed to launch rleInitKernel: " + std::to_string(err));
  }

  // get output locations
  size_t requiredSpace;
  err = hipcub::DeviceScan::ExclusiveSum(
      nullptr, requiredSpace, blockSizes, blockPrefix, grid.x + 1, stream);
  if (err != hipSuccess) {
    throw std::runtime_error(
        "Failed to query rleScanKernel: " + std::to_string(err));
  }

  size_t scanWorkspaceSize
      = std::max(1024 * sizeof(COUNT), maxNum * sizeof(COUNT));
  if (requiredSpace > scanWorkspaceSize) {
    throw std::runtime_error(
        "Too little workspace: " + std::to_string(scanWorkspaceSize) + ", need "
        + std::to_string(requiredSpace));
  }

  err = hipcub::DeviceScan::ExclusiveSum(
      scanWorkspace,
      scanWorkspaceSize,
      blockSizes,
      blockPrefix,
      grid.x + 1,
      stream);
  if (err != hipSuccess) {
    throw std::runtime_error(
        "Failed to launch rleScanKernel: " + std::to_string(err)
        + ", with "
          "grid.x = "
        + std::to_string(grid.x + 1) + " items.");
  }

  // do actual compaction
  rleReduceKernel<VALUE, COUNT, BLOCK_SIZE, GLOBAL_TILE_SIZE>
      <<<grid, block, 0, stream>>>(
          inTyped,
          numInDevice,
          blockPrefix,
          blockStart,
          outValuesTypedPtr,
          outCountsTypedPtr,
          numOutDevice);
  err = hipGetLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(
        "Failed to launch rleReduceKernel: " + std::to_string(err));
  }

  // fix gaps
  rleFinalizeKernel<COUNT, BLOCK_SIZE, GLOBAL_TILE_SIZE>
      <<<dim3(roundUpDiv(grid.x, block.x)), block, 0, stream>>>(
          outCountsTypedPtr, blockStart, blockPrefix, numInDevice);
  err = hipGetLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(
        "Failed to launch rleFinalizeKernel: " + std::to_string(err));
  }
}

} // namespace

/******************************************************************************
 * PUBLIC STATIC FUNCTIONS ****************************************************
 *****************************************************************************/

void RunLengthEncodeGPU::compress(
    void* workspace,
    size_t workspaceSize,
    nvcompType_t valueType,
    void* const outValues,
    nvcompType_t countType,
    void* const outCounts,
    size_t* const numOutDevice,
    const void* const in,
    const size_t num,
    hipStream_t stream)
{
  NVCOMP_TYPE_TWO_SWITCH(
      valueType,
      countType,
      compressInternal,
      workspace,
      workspaceSize,
      outValues,
      outCounts,
      numOutDevice,
      in,
      num,
      stream);
}

void RunLengthEncodeGPU::compressDownstream(
    void* workspace,
    size_t workspaceSize,
    nvcompType_t valueType,
    void** const outValuesPtr,
    nvcompType_t countType,
    void** const outCountsPtr,
    size_t* const numOutDevice,
    const void* const in,
    const size_t* numInDevice,
    const size_t maxNum,
    hipStream_t stream)
{
  NVCOMP_TYPE_TWO_SWITCH(
      valueType,
      countType,
      compressDownstreamInternal,
      workspace,
      workspaceSize,
      outValuesPtr,
      outCountsPtr,
      numOutDevice,
      in,
      numInDevice,
      maxNum,
      stream);
}

size_t RunLengthEncodeGPU::requiredWorkspaceSize(
    const size_t num, const nvcompType_t valueType, const nvcompType_t runType)
{
  NVCOMP_TYPE_TWO_SWITCH_RETURN(
      valueType, runType, requiredWorkspaceSizeTyped, num);
}

} // namespace nvcomp
